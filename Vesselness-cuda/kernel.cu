#include "hip/hip_runtime.h"
// Header files for opencv
#include <iostream> 
#include <time.h>
using namespace std;
#include "opencv2/core/core.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/video/video.hpp"
#include "opencv2/highgui/highgui.hpp"
using namespace cv; 

#include "VesselDetector.h"
#include "Data3D.h"
#include "nstdio.h"
#include "VesselNess.h"
#include "TypeInfo.h"
#include "GLViwerWrapper.h" // For visualization


GLViewerExt viewer;

#include "hip/hip_runtime.h"
#include ""

// files created for this project
#include "ImageProcessing.cuh" 
#include "VesselnessFilter.cuh"
#include "VesselnessFilterPartial.cuh"
#include "VesselDetector.h"



int main()
{
	clock_t t;


	Data3D<short> im_short;
	// bool flag = im_short.load( "../temp/vessel3d.rd.19.data", Vec3i(585,525,105), false, true );
	// bool flag = im_short.load( "../temp/parts/vessel3d.rd.19.part6.data" ); 
	bool flag = im_short.load( "../data/data15.data" ); 
	if(!flag) return 0; 
	// im_short.reset( Vec3i(100,100,100), 500 ); 
	

	//Data3D<int> im_blurred;
	//IPG::GaussianBlur3D( im_short, im_blurred, 5, 1.0 );

	// Computer Vesselness Measure
	Data3D<float> im_float;
	//t = clock();
	//VFG::compute_vesselness( im_short, im_float, 0.9f, 3.6f, 0.2f );
	//t = clock() - t;
	//cout << "It took me " << t << " clicks (" << float(t)/CLOCKS_PER_SEC << " seconds). " << endl; 
	
	// Computer Vesselness Measure
	Data3D<float> im_float2;
	t = clock();
	VFG::compute_vesselness_partial( im_short, im_float2, 2.1f, 2.2f, 0.2f );
	t = clock() - t;
	cout << "It took me " << t << " clicks (" << float(t)/CLOCKS_PER_SEC << " seconds). " << endl; 
	
	//Data3D<short> im_float;
	//IPG::GaussianBlur3D( im_short, im_float, 9, 1.5f );

	//Data3D<Vesselness_All> im_float2;
	//t = clock(); 
	//VF::compute_vesselness( im_short, im_float2, 0.7f, 1.6f, 0.2f );
	//t = clock() - t; 
	//printf ("It took me %d clicks (%f seconds).\n",t,((float)t)/CLOCKS_PER_SEC);
	
	// Visualize result with maximum intensity projection (MIP)
	im_short.remove_margin( 7 ); 
	//im_float.remove_margin( 7 );
	im_float2.remove_margin( 7 ); 
	viewer.addObject( im_short, GLViewer::Volumn::MIP );
	//viewer.addObject( im_float, GLViewer::Volumn::MIP );
	viewer.addObject( im_float2, GLViewer::Volumn::MIP );
	viewer.go(800, 250, 3);

    return 0;
}