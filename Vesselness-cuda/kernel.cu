#include "hip/hip_runtime.h"
// Header files for opencv
#include <iostream> 
#include <time.h>
using namespace std;
#include "opencv2/core/core.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/video/video.hpp"
#include "opencv2/highgui/highgui.hpp"
using namespace cv; 

#include "VesselDetector.h"
#include "Data3D.h"
#include "nstdio.h"
#include "VesselNessTypes.h"
#include "TypeInfo.h"
#include "GLViwerWrapper.h" // For visualization


GLViewerExt viewer;

#include "hip/hip_runtime.h"
#include ""

// files created for this project
#include "ImageProcessing.cuh" 
#include "VesselnessFilter.cuh"
#include "VesselnessFilterPartial.cuh"
#include "VesselDetector.h"


#define INPUT_DIR "./data/"
#define OUTPUT_DIR "../temp/"

int main()
{
	clock_t t;

	bool flag; 
	Image3D<short> im_short;
	flag = im_short.load( "../Vesselness/data/vessel3d.rd.19.data" ); 
	//flag = im_short.load( "../temp/roi20.data" ); 
	//flag = im_short.load( "../temp/roi21.data" ); 
	//flag = im_short.load( "../temp/data/roi16.partial.data" ); 
	if(!flag) return 0; 


	

	// Threshold Data
	//Image3D<short> im_short2;
	//im_short.convertTo( im_short2 );
	IPG::Threshold3D( im_short, im_short, short(2250) );//(1850, 2450)
	


	// Compute Vesselness Measure
	Image3D<float> im_float;
	t = clock();
	VFG::compute_vesselness_partial( im_short, im_float, 
		1.0f, 2.1f, 0.5f,
		1.0e-1f, 5.0f, 3.5e5f,
		300,300,300);
	t = clock() - t;
	cout << "It took me " << t << " clicks (" << float(t)/CLOCKS_PER_SEC << " seconds). " << endl; 
	

	// Visualize result with maximum intensity projection (MIP)
	im_short.shrink_by_half();
	im_float.shrink_by_half();
	im_float.save( "../temp/vessel3d.mmrd.17.vn_float" );
	viewer.addObject( im_float, GLViewer::Volumn::MIP );
	viewer.addObject( im_short, GLViewer::Volumn::MIP );
	
	
	viewer.go(600, 300, 2);

    return 0;
}
