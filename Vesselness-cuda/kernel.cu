#include "hip/hip_runtime.h"
// Header files for opencv
#include <iostream> 
#include <time.h>
using namespace std;
#include "opencv2/core/core.hpp"
#include "opencv2/imgproc/imgproc.hpp"
#include "opencv2/video/video.hpp"
#include "opencv2/highgui/highgui.hpp"
using namespace cv; 

#include "VesselDetector.h"
#include "Data3D.h"
#include "nstdio.h"
#include "VesselNess.h"
#include "TypeInfo.h"
#include "GLViwerWrapper.h" // For visualization


GLViewerExt viewer;

#include "hip/hip_runtime.h"
#include ""

// files created for this project
#include "ImageProcessing.cuh" 
#include "VesselnessFilter.cuh"
#include "VesselDetector.h"


int main()
{
	clock_t t;

	Data3D<short> im_short;
	bool flag = im_short.load( "../data/data15.data" );
	if(!flag) return 0; 
	
	Data3D<int> im_blurred;
	IPG::GaussianBlur3D( im_short, im_blurred, 5, 1.0 );

	// Computer Vesselness Measure
	Data3D<float> im_float;
	t = clock();
	VFG::compute_vesselness( im_short, im_float, 0.7f, 1.6f, 0.2f );
	t = clock() - t;
	printf ("It took me %d clicks (%f seconds).\n",t,((float)t)/CLOCKS_PER_SEC);
	
	Data3D<Vesselness_All> im_float2;
	t = clock(); 
	VF::compute_vesselness( im_short, im_float2, 0.7f, 1.6f, 0.2f );
	t = clock() - t; 
	printf ("It took me %d clicks (%f seconds).\n",t,((float)t)/CLOCKS_PER_SEC);
	
	// Visualize result with maximum intensity projection (MIP)
	im_short.remove_margin( 7 ); 
	im_float.remove_margin( 7 );
	im_float2.remove_margin( 7 ); 
	viewer.addObject( im_short, GLViewer::Volumn::MIP );
	viewer.addObject( im_float, GLViewer::Volumn::MIP );
	viewer.addObject( im_float2, GLViewer::Volumn::MIP );
	viewer.go(600, 200, 3);





    return 0;
}